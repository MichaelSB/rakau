#include "hip/hip_runtime.h"
#include <array>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <memory>
#include <new>
#include <vector>

#include <boost/numeric/conversion/cast.hpp>
#include <boost/preprocessor/seq/elem.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>

#include <rakau/detail/tree_fwd.hpp>

namespace rakau
{

inline namespace detail
{

// Minimum number of particles needed for running the cuda implementation.
unsigned cuda_min_size()
{
    return 1000u;
}

// Get the number of cuda devices.
unsigned cuda_device_count()
{
    int ret;
    if (::hipGetDeviceCount(&ret) != ::hipSuccess) {
        throw std::runtime_error("Cannot determine the number of CUDA devices");
    }
    return static_cast<unsigned>(ret);
}

// Small helper to create a unique_ptr to managed memory
// with enough storage for n objects of type T.
template <typename T>
auto make_scoped_cu_array(std::size_t n)
{
    if (n > std::numeric_limits<std::size_t>::max() / sizeof(T)) {
        throw std::bad_alloc{};
    }
    void *ret;
    if (::hipMallocManaged(&ret, n * sizeof(T)) != ::hipSuccess) {
        throw std::bad_alloc{};
    }
    return std::unique_ptr<T, decltype(::hipFree) *>(static_cast<T *>(ret), ::hipFree);
}

// Small wrapper to handle arrays in managed memory.
template <typename T>
class scoped_cu_array
{
    using ptr_t = decltype(make_scoped_cu_array<T>(0));

public:
    // Def ctor, inits to nullptr.
    scoped_cu_array() : m_ptr(nullptr, ::hipFree) {}
    // Constructor from size.
    explicit scoped_cu_array(std::size_t n) : m_ptr(make_scoped_cu_array<T>(n)) {}
    // Get a pointer to the start of the array.
    T *get() const
    {
        return m_ptr.get();
    }

private:
    ptr_t m_ptr;
};

template <unsigned Q, std::size_t NDim, typename F, typename UInt>
__global__ void acc_pot_kernel(F *__restrict__ res_x, F *__restrict__ res_y, F *__restrict__ res_z,
                               const F *__restrict__ ptr_x, const F *__restrict__ ptr_y, const F *__restrict__ ptr_z,
                               const F *__restrict__ ptr_mass, const UInt *__restrict__ codes, const int nparts,
                               const tree_node_t<NDim, F, UInt> *__restrict__ tree_ptr, const int tree_size,
                               const F theta2, const F G, const F eps2)
{
    int pidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pidx >= nparts) {
        // Don't do anything if we are in the remainder
        // of the last tile.
        return;
    }

    // Results, inited to zero.
    F r_x = 0, r_y = 0, r_z = 0;

    // Load the particle code and position.
    const auto code = codes[pidx];
    const auto x = ptr_x[pidx], y = ptr_y[pidx], z = ptr_z[pidx];

    const auto s_p_code_init = static_cast<UInt>(code | (UInt(1) << (cbits_v<UInt, NDim> * NDim)));

    for (auto src_idx = 0; src_idx < tree_size;) {
        const auto &src_node = tree_ptr[src_idx];
        const auto src_code = src_node.code;
        const auto src_begin = static_cast<int>(src_node.begin), src_end = static_cast<int>(src_node.end);
        const auto n_children_src = static_cast<int>(src_node.n_children);
        auto node_x = src_node.props[0], node_y = src_node.props[1], node_z = src_node.props[2],
             node_mass = src_node.props[3];
        const auto src_level = src_node.level;
        const auto src_dim2 = src_node.dim2;

        const auto s_p_code = s_p_code_init >> ((cbits_v<UInt, NDim> - src_level) * NDim);

        F diff_x = node_x - x, diff_y = node_y - y, diff_z = node_z - z,
          dist2 = diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;
        if (__all_sync(unsigned(-1), s_p_code != src_code && src_dim2 < theta2 * dist2)) {
            dist2 += eps2;
            const auto dist = sqrt(dist2);
            const auto node_mass_dist3 = node_mass / (dist * dist2);
            r_x += diff_x * node_mass_dist3;
            r_y += diff_y * node_mass_dist3;
            r_z += diff_z * node_mass_dist3;
            src_idx += n_children_src + 1;
        } else {
            if (!n_children_src) {
                for (auto i = src_begin; i < src_end; ++i) {
                    const bool is_tgt_particle = pidx == i;
                    dist2 = eps2 + is_tgt_particle;
                    diff_x = ptr_x[i] - x;
                    diff_y = ptr_y[i] - y;
                    diff_z = ptr_z[i] - z;
                    dist2 += diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;
                    const auto dist = sqrt(dist2), m_i = ptr_mass[i];
                    const auto m_i_dist3 = m_i / (dist * dist2);
                    r_x += diff_x * m_i_dist3;
                    r_y += diff_y * m_i_dist3;
                    r_z += diff_z * m_i_dist3;
                }
            }
            ++src_idx;
        }
    }

    res_x[pidx] = G * r_x;
    res_y[pidx] = G * r_y;
    res_z[pidx] = G * r_z;
} // namespace detail

template <unsigned Q, std::size_t NDim, typename F, typename UInt>
void cuda_acc_pot_impl(const std::array<F *, tree_nvecs_res<Q, NDim>> &out,
                       const std::vector<tree_size_t<F>> &split_indices, const tree_node_t<NDim, F, UInt> *tree,
                       tree_size_t<F> tree_size, const std::array<const F *, NDim + 1u> &p_parts, const UInt *codes,
                       tree_size_t<F> nparts, F theta2, F G, F eps2)
{
    // TODO error handling for memcopy.
    // TODO numeric casting?

    // Create the arrays for the results.
    std::array<scoped_cu_array<F>, tree_nvecs_res<Q, NDim>> res_arrs;
    F *res_ptrs[tree_nvecs_res<Q, NDim>];
    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        res_arrs[j] = scoped_cu_array<F>(boost::numeric_cast<std::size_t>(nparts));
        res_ptrs[j] = res_arrs[j].get();
    }

    scoped_cu_array<tree_node_t<NDim, F, UInt>> tree_arr(boost::numeric_cast<std::size_t>(tree_size));
    ::hipMemcpy(tree_arr.get(), tree, sizeof(tree_node_t<NDim, F, UInt>) * tree_size, ::hipMemcpyHostToDevice);
    const tree_node_t<NDim, F, UInt> *tree_ptr = tree_arr.get();

    std::array<scoped_cu_array<F>, NDim + 1u> parts_arrs;
    const F *parts_ptrs[NDim + 1u];
    for (std::size_t j = 0; j < NDim + 1u; ++j) {
        parts_arrs[j] = scoped_cu_array<F>(boost::numeric_cast<std::size_t>(nparts));
        ::hipMemcpy(parts_arrs[j].get(), p_parts[j], sizeof(F) * nparts, ::hipMemcpyHostToDevice);
        parts_ptrs[j] = parts_arrs[j].get();
    }

    scoped_cu_array<UInt> codes_arr(boost::numeric_cast<std::size_t>(nparts));
    ::hipMemcpy(codes_arr.get(), codes, sizeof(UInt) * nparts, ::hipMemcpyHostToDevice);
    const UInt *codes_ptr = codes_arr.get();

    // TODO overflow checks on ints?
    acc_pot_kernel<Q, NDim, F, UInt><<<(nparts + 31u) / 32u, 32u>>>(
        res_ptrs[0], res_ptrs[1], res_ptrs[2], parts_ptrs[0], parts_ptrs[1], parts_ptrs[2], parts_ptrs[3], codes_ptr,
        boost::numeric_cast<int>(nparts), tree_ptr, boost::numeric_cast<int>(tree_size), theta2, G, eps2);

    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        ::hipMemcpy(out[j], res_ptrs[j], sizeof(F) * nparts, ::hipMemcpyDeviceToHost);
    }
}

// Explicit instantiations of the templates implemented above. We are going to use Boost.Preprocessor.
// It's gonna look ugly, but it will allow us to avoid a lot of typing.

// Define the values/types that we will use for the concrete instantiations.

// Only quadtrees and octrees for the moment.
#define RAKAU_CUDA_INST_DIM_SEQUENCE (2)(3)

// float and double only on the gpu.
#define RAKAU_CUDA_INST_FP_SEQUENCE (float)(double)

// 32/64bit types for the particle codes.
#define RAKAU_CUDA_INST_UINT_SEQUENCE (std::uint32_t)(std::uint64_t)

// Computation of accelerations, potentials or both.
#define RAKAU_CUDA_INST_Q_SEQUENCE (0)(1)(2)

// Macro for the instantiation of the main function. NDim, F, UInt and Q will be passed in
// as a sequence named Args (in that order).
#define RAKAU_CUDA_EXPLICIT_INST_FUN(r, Args)                                                                          \
    template void cuda_acc_pot_impl<BOOST_PP_SEQ_ELEM(3, Args), BOOST_PP_SEQ_ELEM(0, Args),                            \
                                    BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(2, Args)>(                           \
        const std::array<BOOST_PP_SEQ_ELEM(1, Args) *,                                                                 \
                         tree_nvecs_res<BOOST_PP_SEQ_ELEM(3, Args), BOOST_PP_SEQ_ELEM(0, Args)>> &,                    \
        const std::vector<tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>> &,                                                  \
        const tree_node_t<BOOST_PP_SEQ_ELEM(0, Args), BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(2, Args)> *,       \
        tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>,                                                                       \
        const std::array<const BOOST_PP_SEQ_ELEM(1, Args) *, BOOST_PP_SEQ_ELEM(0, Args) + 1u> &,                       \
        const BOOST_PP_SEQ_ELEM(2, Args) *, tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>, BOOST_PP_SEQ_ELEM(1, Args),       \
        BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(1, Args));

// Do the actual instantiation via a cartesian product over the sequences.
// clang-format off
BOOST_PP_SEQ_FOR_EACH_PRODUCT(RAKAU_CUDA_EXPLICIT_INST_FUN, (RAKAU_CUDA_INST_DIM_SEQUENCE)(RAKAU_CUDA_INST_FP_SEQUENCE)(RAKAU_CUDA_INST_UINT_SEQUENCE)(RAKAU_CUDA_INST_Q_SEQUENCE));
// clang-format on

} // namespace detail
} // namespace rakau
