#include "hip/hip_runtime.h"
#include <array>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <memory>
#include <new>

#include <boost/numeric/conversion/cast.hpp>

#include <rakau/detail/hc_fwd.hpp>
#include <rakau/detail/tree_fwd.hpp>

namespace rakau
{

inline namespace detail
{

template <typename T>
auto make_scoped_cu_array(std::size_t n)
{
    if (n > std::numeric_limits<std::size_t>::max() / sizeof(T)) {
        throw std::bad_alloc();
    }
    T *ret;
    auto res = ::hipMalloc(&ret, n * sizeof(T));
    if (res != ::hipSuccess) {
        throw std::bad_alloc();
    }
    return std::unique_ptr<T, decltype(::hipFree) *>(ret, ::hipFree);
}

template <typename T>
class scoped_cu_array
{
    using ptr_t = decltype(make_scoped_cu_array<T>(0));

public:
    scoped_cu_array() : m_ptr(nullptr, ::hipFree) {}
    explicit scoped_cu_array(std::size_t n) : m_ptr(make_scoped_cu_array<T>(n)) {}
    auto get() const
    {
        return m_ptr.get();
    }

private:
    ptr_t m_ptr;
};

template <unsigned Q, std::size_t NDim, typename F, typename UInt>
__global__ void acc_pot_kernel(F *(&res_ptrs)[tree_nvecs_res<Q, NDim>], const F *(&parts_ptrs)[NDim + 1u],
                               const UInt *codes, const int nparts, const tree_node_t<NDim, F, UInt> *tree_ptr,
                               const int tree_size, const F theta2, const F G, const F eps2)
{
    int pidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pidx >= nparts) {
        // Don't do anything if we are in the remainder
        // of the last tile.
        return;
    }

    // Array of results, inited to zeroes.
    F res_array[sizeof(res_ptrs) / sizeof(F *)]{};

    // Load the particle code, position and mass.
    const auto p_code = codes[pidx];
    F p_pos[NDim];
    for (std::size_t j = 0; j < NDim; ++j) {
        p_pos[j] = parts_ptrs[j][pidx];
    }
    const auto p_mass = parts_ptrs[NDim][pidx];

    // Temporary arrays that will be used in the loop.
    F dist_vec[NDim], props[NDim + 1u];

    // Add a 1 bit just above the highest possible bit position for the particle code.
    // This value is used in the loop, we precompute it here.
    const auto s_p_code_init = static_cast<UInt>(p_code | (UInt(1) << (cbits_v<UInt, NDim> * NDim)));

    // Loop over the tree.
    for (auto src_idx = 0; src_idx < tree_size;) {
        // Get a reference to the current source node, and cache locally a few quantities.
        const auto &src_node = tree_ptr[src_idx];
        // Code of the source node.
        const auto src_code = src_node.code;
        // Range of the source node.
        const auto src_begin = static_cast<int>(src_node.begin), src_end = static_cast<int>(src_node.end);
        // Number of children of the source node.
        const auto n_children_src = static_cast<int>(src_node.n_children);
        // Node properties.
        for (std::size_t j = 0; j < NDim + 1u; ++j) {
            props[j] = src_node.props[j];
        }
        // Level of the source node.
        const auto src_level = src_node.level;
        // Square of the dimension of the source node.
        const auto src_dim2 = src_node.dim2;

        // Compute the shifted particle code. This is the particle code with one extra
        // top bit and then shifted down according to the level of the source node, so that
        // the top 1 bits of s_p_code and src_code are at the same position.
        const auto s_p_code = static_cast<UInt>(s_p_code_init >> ((cbits_v<UInt, NDim> - src_level) * NDim));
        // We need now to determine if the source node contains the target particle.
        // If it does, in all but one specific case we will have to correct
        // the source node COM coordinates & mass with the removal of the target particle.
        // The only exception is if the source node contains *only* the target particle,
        // in which case we want to avoid the correction because it would result in
        // infinities.
        //
        // The check s_p_code == src_code tells us if the source node contains the target
        // particle. The check (src_end - src_begin) != 1 tells us that the source node
        // contains other particles in addition to the target particle.
        if (s_p_code == src_code && (src_end - src_begin) != 1) {
            // Update the COM position.
            const auto new_node_mass = props[NDim] - p_mass;
            for (std::size_t j = 0; j < NDim; ++j) {
                props[j] = (props[j] * props[NDim] - p_mass * p_pos[j]) / new_node_mass;
            }
            // Don't forget to update the node mass as well.
            props[NDim] = new_node_mass;
        }

        // Compute the distance between target particle and source COM.
        // NOTE: if we are in a source node which contains only the target particle,
        // then dist2 and dist_vec will be zero.
        F dist2(0);
        for (std::size_t j = 0; j < NDim; ++j) {
            const auto diff = props[j] - p_pos[j];
            dist2 += diff * diff;
            dist_vec[j] = diff;
        }
        // Now let's run the BH check on *all* the target particles in the same wavefront.
        // NOTE: if we are in a source node which contains only the target particle,
        // then dist2 will have been set to zero above and the check always fails.
        if (__all_sync(unsigned(-1), src_dim2 < theta2 * dist2)) {
            // We are not in a leaf node containing only the target particle,
            // and the source node satisfies the BH criterion for the target
            // particle. We will then add the (approximated) contribution of the source node
            // to the final result.
            //
            // Start by adding the softening.
            dist2 += eps2;
            // Compute the (softened) distance.
            const auto dist = sqrt(dist2);
            const auto node_mass_dist3 = props[NDim] / (dist * dist2);
            for (std::size_t j = 0; j < NDim; ++j) {
                res_array[j] += dist_vec[j] * node_mass_dist3;
            }
            // We can now skip all the children of the source node.
            src_idx += n_children_src + 1;
        } else {
            // Either the source node fails the BH criterion, or we are in a source node which
            // contains only the target particle.
            if (!n_children_src) {
                // We are in a leaf node. Compute all the interactions with the target particle.
                // NOTE: if we are in a source node which contains only the target particle,
                // then the loop will have just 1 iteration and the use of the is_tgt_particle
                // variable will ensure that all interactions of the particle with itself
                // amount to zero.
                for (auto i = src_begin; i < src_end; ++i) {
                    // Test if the current particle of the source leaf node coincides
                    // with the target particle.
                    const bool is_tgt_particle = pidx == i;
                    // Init the distance with the softening, plus add some extra
                    // softening if i is the target particle. This will avoid
                    // infinites when dividing by the distance below.
                    dist2 = eps2 + is_tgt_particle;
                    for (std::size_t j = 0; j < NDim; ++j) {
                        const auto diff = parts_ptrs[j][i] - p_pos[j];
                        dist2 += diff * diff;
                        dist_vec[j] = diff;
                    }
                    // Compute the distance, load the current source mass.
                    const auto dist = sqrt(dist2), m_i = parts_ptrs[NDim][i];
                    // Q == 0 or 2: accelerations are requested.
                    const auto m_i_dist3 = m_i / (dist * dist2);
                    for (std::size_t j = 0; j < NDim; ++j) {
                        // NOTE: if i == pidx, then dist_vec will be a vector
                        // of zeroes and res_array will not be modified.
                        res_array[j] += dist_vec[j] * m_i_dist3;
                    }
                }
            }
            // In any case, we keep traversing the tree moving to the next node
            // in depth-first order.
            ++src_idx;
        }
    }

    // Handle the G constant and write out the result.
    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        res_ptrs[j][pidx] = G * res_array[j];
    }
} // namespace detail

template <unsigned Q, std::size_t NDim, typename F, typename UInt>
void acc_pot_impl_cuda(const std::array<F *, tree_nvecs_res<Q, NDim>> &out, const tree_node_t<NDim, F, UInt> *tree,
                       tree_size_t<F> tree_size, const std::array<const F *, NDim + 1u> &p_parts, const UInt *codes,
                       tree_size_t<F> nparts, F theta2, F G, F eps2, tree_size_t<F> ncrit)
{
    static_assert(Q == 0u);

    // TODO error handling for memcopy.
    std::array<scoped_cu_array<F>, tree_nvecs_res<Q, NDim>> res_arrs;
    F *res_ptrs[tree_nvecs_res<Q, NDim>];
    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        res_arrs[j] = scoped_cu_array<F>(boost::numeric_cast<std::size_t>(nparts));
        res_ptrs[j] = res_arrs[j].get();
    }

    scoped_cu_array<tree_node_t<NDim, F, UInt>> tree_arr(boost::numeric_cast<std::size_t>(tree_size));
    ::hipMemcpy(tree_arr.get(), tree, sizeof(tree_node_t<NDim, F, UInt>) * tree_size, ::hipMemcpyHostToDevice);
    const tree_node_t<NDim, F, UInt> *tree_ptr = tree_arr.get();

    std::array<scoped_cu_array<F>, NDim + 1u> parts_arrs;
    const F *parts_ptrs[NDim + 1u];
    for (std::size_t j = 0; j < NDim + 1u; ++j) {
        parts_arrs[j] = scoped_cu_array<F>(boost::numeric_cast<std::size_t>(nparts));
        ::hipMemcpy(parts_arrs[j].get(), p_parts[j], sizeof(F) * nparts, ::hipMemcpyHostToDevice);
        parts_ptrs[j] = parts_arrs[j].get();
    }

    scoped_cu_array<UInt> codes_arr(boost::numeric_cast<std::size_t>(nparts));
    ::hipMemcpy(codes_arr.get(), codes, sizeof(UInt) * nparts, ::hipMemcpyHostToDevice);
    const UInt *codes_ptr = codes_arr.get();

    // TODO overflow checks on ints?
    acc_pot_kernel<Q, NDim, F, UInt>
        <<<(nparts + 31u) / 32u, 32u>>>(res_ptrs, parts_ptrs, codes_ptr, boost::numeric_cast<int>(nparts), tree_ptr,
                                        boost::numeric_cast<int>(tree_size), theta2, G, eps2);

    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        ::hipMemcpy(res_ptrs[j], out[j], sizeof(F) * nparts, ::hipMemcpyDeviceToHost);
    }
}

// Explicit instantiations.
#define RAKAU_CUDA_EXPLICIT_INST(Q, NDim, F, UInt)                                                                     \
    template void acc_pot_impl_cuda<Q, NDim, F, UInt>(                                                                 \
        const std::array<F *, tree_nvecs_res<Q, NDim>> &, const tree_node_t<NDim, F, UInt> *, tree_size_t<F>,          \
        const std::array<const F *, NDim + 1u> &, const UInt *, tree_size_t<F>, F, F, F, tree_size_t<F>)

RAKAU_CUDA_EXPLICIT_INST(0, 3, float, std::uint64_t);

RAKAU_CUDA_EXPLICIT_INST(0, 3, double, std::uint64_t);

RAKAU_CUDA_EXPLICIT_INST(0, 3, float, std::uint32_t);

RAKAU_CUDA_EXPLICIT_INST(0, 3, double, std::uint32_t);

#undef RAKAU_CUDA_EXPLICIT_INST

} // namespace detail
} // namespace rakau
