#include "hip/hip_runtime.h"
#include <array>
#include <cassert>
#include <cstddef>
#include <cstdint>
#include <limits>
#include <memory>
#include <new>
#include <stdexcept>
#include <utility>
#include <vector>

#include <boost/numeric/conversion/cast.hpp>
#include <boost/preprocessor/seq/elem.hpp>
#include <boost/preprocessor/seq/for_each_product.hpp>

#include <rakau/detail/tree_fwd.hpp>

namespace rakau
{

inline namespace detail
{

// Minimum number of particles needed for running the cuda implementation.
unsigned cuda_min_size()
{
    return 1000u;
}

// Get the number of cuda devices.
unsigned cuda_device_count()
{
    int ret;
    if (::hipGetDeviceCount(&ret) != ::hipSuccess) {
        throw std::runtime_error("Cannot determine the number of CUDA devices");
    }
    return static_cast<unsigned>(ret);
}

// Small helper to create a unique_ptr to managed memory
// with enough storage for n objects of type T.
template <typename T>
auto make_scoped_cu_array(std::size_t n)
{
    if (n > std::numeric_limits<std::size_t>::max() / sizeof(T)) {
        throw std::bad_alloc{};
    }
    void *ret;
    if (::hipMallocManaged(&ret, n * sizeof(T)) != ::hipSuccess) {
        throw std::bad_alloc{};
    }
    return std::unique_ptr<T, decltype(::hipFree) *>(static_cast<T *>(ret), ::hipFree);
}

// Small wrapper to create and manage an array of objects of type T
// in CUDA managed memory.
// NOTE: we need this instead of a naked unique_ptr because we want
// to be able to default-construct for use in arrays.
template <typename T>
class scoped_cu_array
{
    using ptr_t = decltype(make_scoped_cu_array<T>(0));

public:
    // Def ctor, inits to nullptr.
    scoped_cu_array() : m_ptr(nullptr, ::hipFree) {}
    // Constructor from size.
    explicit scoped_cu_array(std::size_t n) : m_ptr(make_scoped_cu_array<T>(n)) {}
    // Get a pointer to the start of the array.
    T *get() const
    {
        return m_ptr.get();
    }

private:
    ptr_t m_ptr;
};

// A few CUDA API wrappers with some minimal error checking.

static inline void cuda_memcpy(void *dst, const void *src, std::size_t count, ::hipMemcpyKind kind)
{
    if (::hipMemcpy(dst, src, count, kind) != ::hipSuccess) {
        throw std::runtime_error("hipMemcpy() returned an error code");
    }
}

static inline void cuda_memcpy_async(void *dst, const void *src, std::size_t count, ::hipMemcpyKind kind,
                                     ::hipStream_t stream)
{
    if (::hipMemcpyAsync(dst, src, count, kind, stream) != ::hipSuccess) {
        throw std::runtime_error("hipMemcpyAsync() returned an error code");
    }
}

static inline void cuda_device_synchronize()
{
    if (::hipDeviceSynchronize() != ::hipSuccess) {
        throw std::runtime_error("hipDeviceSynchronize() returned an error code");
    }
}

static inline void cuda_set_device(int device)
{
    if (::hipSetDevice(device) != ::hipSuccess) {
        throw std::runtime_error("hipSetDevice() returned an error code");
    }
}

static inline void cuda_mem_advise(const void *ptr, std::size_t count, ::hipMemoryAdvise advice, int device)
{
    const auto ret = ::hipMemAdvise(ptr, count, advice, device);
    // NOTE: this might fail with hipErrorInvalidDevice on older
    // devices, in which case we don't want to error out.
    if (ret != ::hipSuccess && ret != ::hipErrorInvalidDevice) {
        throw std::runtime_error("hipMemAdvise() returned an error code");
    }
}

static inline void cuda_stream_create(::hipStream_t *ptr)
{
    if (::hipStreamCreate(ptr) != ::hipSuccess) {
        throw std::runtime_error("hipStreamCreate() returned an error code");
    }
}

// Small wrapper to allow passing arrays of pointers
// as kernel arguments.
template <typename T, std::size_t N>
struct arr_wrap {
    T value[N];
};

// Kernel for the computation of accelerations/potentials.
template <unsigned Q, std::size_t NDim, typename F, typename UInt>
__global__ void acc_pot_kernel(arr_wrap<F *, tree_nvecs_res<Q, NDim>> res_ptrs, int p_begin, int p_end,
                               const tree_node_t<NDim, F, UInt> *tree_ptr, int tree_size,
                               arr_wrap<const F *, NDim + 1u> parts_ptrs, const UInt *codes_ptr, F theta2, F G, F eps2)
{
    // Get the local and global particle indices.
    const auto loc_idx = blockIdx.x * blockDim.x + threadIdx.x;
    const auto pidx = loc_idx + p_begin;
    if (pidx >= p_end) {
        // Don't do anything if we are in the remainder
        // of the last block.
        return;
    }

    // Array of results, inited to zeroes.
    constexpr auto res_array_size = tree_nvecs_res<Q, NDim>;
    F res_array[res_array_size]{};

    // Load the particle code, position and mass.
    const auto p_code = codes_ptr[pidx];
    F p_pos[NDim];
    for (std::size_t j = 0; j < NDim; ++j) {
        p_pos[j] = parts_ptrs.value[j][pidx];
    }
    const auto p_mass = parts_ptrs.value[NDim][pidx];

    // Temporary arrays that will be used in the loop.
    F dist_vec[NDim], props[NDim + 1u];

    // Add a 1 bit just above the highest possible bit position for the particle code.
    // This value is used in the loop, we precompute it here.
    const auto s_p_code_init = static_cast<UInt>(p_code | (UInt(1) << (cbits_v<UInt, NDim> * NDim)));

    // Loop over the tree.
    for (auto src_idx = 0; src_idx < tree_size;) {
        // Get a reference to the current source node, and cache locally a few quantities.
        const auto &src_node = tree_ptr[src_idx];
        // Code of the source node.
        const auto src_code = src_node.code;
        // Range of the source node.
        const auto src_begin = static_cast<int>(src_node.begin), src_end = static_cast<int>(src_node.end);
        // Number of children of the source node.
        const auto n_children_src = static_cast<int>(src_node.n_children);
        // Node properties.
        for (std::size_t j = 0; j < NDim + 1u; ++j) {
            props[j] = src_node.props[j];
        }
        // Level of the source node.
        const auto src_level = src_node.level;
        // Square of the dimension of the source node.
        const auto src_dim2 = src_node.dim2;

        // Compute the shifted particle code. This is the particle code with one extra
        // top bit and then shifted down according to the level of the source node, so that
        // the top 1 bits of s_p_code and src_code are at the same position.
        // If s_p_code == src_code, then it means that the source node contains the target particle
        // (or, in other words, the source node is an ancestor of the leaf node containing
        // the target particle).
        const auto s_p_code = s_p_code_init >> ((cbits_v<UInt, NDim> - src_level) * NDim);
        // If the source node contains the target particle, we will need to account for self interactions
        // in the tree traversal. There are two different approaches that can be taken.
        //
        // The first is to modify on-the-fly the properties of the source node with the removal of the target
        // particle. In the classic BH scheme, this will alter the COM position of the source node
        // and its mass. The alteration needs to take place if the source node is an ancestor of
        // NT, the leaf node of the target particle. However, if the source node coincides with NT
        // and NT contains *only* the target particle, then the alteration must not take place because
        // otherwise we generate infinities (the COM of a system of only 1 particle is the particle
        // itself). The alteration can be done by defining a mass factor mf as
        //
        // mf = orig_node_mass / (orig_node_mass - p_mass * needs_alteration),
        //
        // where needs_alteration is a boolean that expresses whether the source node needs to be
        // adjusted or not (so that mf == 1 if no adjustment needs to happen). It can then be shown
        // that the target particle's distance from the adjusted COM is
        //
        // new_dist = mf * orig_dist,
        //
        // where orig_dist is the (vector) distance from the original COM. The new node mass will be:
        //
        // new_node_mass = orig_node_mass - p_mass * needs_alteration.
        //
        // The other approach is not to modify the properties of the COM, and instead just continue
        // in the tree traversal as if the current source node didn't satisfy the BH check. By doing this
        // we will eventually land into the leaf node of the target particle, where we will compute
        // local particle-particle interactions in the usual N**2 way (avoiding self interactions for the
        // target particle).
        //
        // The first method is more arithmetically-intensive and requires less flow control. The other
        // method will result in longer tree traversals and higher flow control, but requires less arithmetics.
        // At the moment it seems like the first method might be a bit faster on the GPU, but it's also not
        // entirely clear how more complicated/intensive the source node alteration would become once we
        // implement quadrupole moments and other MACs. Thus, for now, let's go with the second approach.

        // Compute the distance between target particle and source COM.
        // NOTE: if we are in a source node which contains only the target particle,
        // then dist2 and dist_vec will be zero.
        F dist2(0);
        for (std::size_t j = 0; j < NDim; ++j) {
            const auto diff = props[j] - p_pos[j];
            dist2 += diff * diff;
            dist_vec[j] = diff;
        }

        // Now let's run the BH/ancestor check on all the target particles in the same warp.
        if (__all_sync(unsigned(-1), s_p_code != src_code && src_dim2 < theta2 * dist2)) {
            // The source node does not contain the target particle and it satisfies the BH check.
            // We will then add the (approximated) contribution of the source node
            // to the final result.
            //
            // Start by adding the softening.
            dist2 += eps2;
            // Compute the (softened) distance.
            const auto dist = sqrt(dist2);
            if (Q == 0u || Q == 2u) {
                // Q == 0 or 2: accelerations are requested.
                const auto node_mass_dist3 = props[NDim] / (dist * dist2);
                for (std::size_t j = 0; j < NDim; ++j) {
                    res_array[j] += dist_vec[j] * node_mass_dist3;
                }
            }
            if (Q == 1u || Q == 2u) {
                // Q == 1 or 2: potentials are requested.
                // Establish the index of the potential in the result array:
                // 0 if only the potentials are requested, NDim otherwise.
                constexpr auto pot_idx = static_cast<std::size_t>(Q == 1u ? 0u : NDim);
                // Add the potential due to the node.
                res_array[pot_idx] -= p_mass * props[NDim] / dist;
            }
            // We can now skip all the children of the source node.
            src_idx += n_children_src + 1;
        } else {
            // Either the source node contains the target particle, or it fails the BH check.
            if (!n_children_src) {
                // We are in a leaf node (possibly containing the target particle).
                // Compute all the interactions with the target particle.
                for (auto i = src_begin; i < src_end; ++i) {
                    // Test if the current particle of the source leaf node coincides
                    // with the target particle.
                    const bool is_tgt_particle = pidx == i;
                    // Init the distance with the softening, plus add some extra
                    // softening if i is the target particle. This will avoid
                    // infinites when dividing by the distance below.
                    dist2 = eps2 + is_tgt_particle;
                    for (std::size_t j = 0; j < NDim; ++j) {
                        const auto diff = parts_ptrs.value[j][i] - p_pos[j];
                        dist2 += diff * diff;
                        dist_vec[j] = diff;
                    }
                    // Compute the distance, load the current source mass.
                    const auto dist = sqrt(dist2), m_i = parts_ptrs.value[NDim][i];
                    if (Q == 0u || Q == 2u) {
                        // Q == 0 or 2: accelerations are requested.
                        const auto m_i_dist3 = m_i / (dist * dist2);
                        for (std::size_t j = 0; j < NDim; ++j) {
                            // NOTE: if i == pidx, then dist_vec will be a vector
                            // of zeroes and res_array will not be modified.
                            res_array[j] += dist_vec[j] * m_i_dist3;
                        }
                    }
                    if (Q == 1u || Q == 2u) {
                        // Q == 1 or 2: potentials are requested.
                        // Establish the index of the potential in the result array:
                        // 0 if only the potentials are requested, NDim otherwise.
                        constexpr auto pot_idx = static_cast<std::size_t>(Q == 1u ? 0u : NDim);
                        // NOTE: for the potential, we need an extra multiplication by
                        // !is_tgt_particle to set the rhs to zero in case i == pidx (for
                        // the accelerations, the same effect was achieved via dist_vec).
                        res_array[pot_idx] -= !is_tgt_particle * p_mass * m_i / dist;
                    }
                }
            }
            // Keep traversing the tree moving to the next node in depth-first order.
            ++src_idx;
        }
    }

    // Handle the G constant and write out the result.
    for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
        // NOTE: for writing the results, we use the local index.
        res_ptrs.value[j][loc_idx] = G * res_array[j];
    }
}

// Small wrapper to automatically execute a function upon destruction.
template <typename T>
struct scoped_guard {
    explicit scoped_guard(const T &f) : m_f(f) {}
    ~scoped_guard()
    {
        m_f();
    }
    const T &m_f;
};

template <unsigned Q, std::size_t NDim, typename F, typename UInt>
void cuda_acc_pot_impl(const std::array<F *, tree_nvecs_res<Q, NDim>> &out,
                       const std::vector<tree_size_t<F>> &split_indices, const tree_node_t<NDim, F, UInt> *tree,
                       tree_size_t<F> tree_size, const std::array<const F *, NDim + 1u> &p_parts, const UInt *codes,
                       tree_size_t<F> nparts, F theta2, F G, F eps2)
{
    assert(split_indices.size() && split_indices.size() - 1u <= cuda_device_count());

    // Attempt to pin the input and output memory. We will record
    // in an array of booleans which areas were actually successfully
    // pinned, so that we can unpin them later.
    std::array<bool, tree_nvecs_res<Q, NDim> + NDim + 3u> pin_flags{};
    auto pin_memory = [&pin_flags, &out, nparts, tree, tree_size, codes, &p_parts]() {
        std::size_t i = 0;
        for (auto ptr : out) {
            pin_flags[i++]
                = ::hipHostRegister(const_cast<void *>((const void *)ptr), sizeof(F) * nparts, hipHostRegisterDefault)
                  == ::hipSuccess;
        }
        pin_flags[i++] = ::hipHostRegister(const_cast<void *>((const void *)tree),
                                            sizeof(tree_node_t<NDim, F, UInt>) * tree_size, hipHostRegisterDefault)
                         == ::hipSuccess;
        pin_flags[i++] = ::hipHostRegister(const_cast<void *>((const void *)codes), sizeof(UInt) * nparts,
                                            hipHostRegisterDefault)
                         == ::hipSuccess;
        for (auto ptr : p_parts) {
            pin_flags[i++]
                = ::hipHostRegister(const_cast<void *>((const void *)ptr), sizeof(F) * nparts, hipHostRegisterDefault)
                  == ::hipSuccess;
        }
    };
    pin_memory();

    // Function to unpin the pinned memory areas.
    auto unpin_memory = [&pin_flags, &out, tree, codes, &p_parts]() {
        std::size_t i = 0;
        for (auto ptr : out) {
            if (pin_flags[i++]) {
                ::hipHostUnregister(const_cast<void *>((const void *)ptr));
            }
        }
        if (pin_flags[i++]) {
            ::hipHostUnregister(const_cast<void *>((const void *)tree));
        }
        if (pin_flags[i++]) {
            ::hipHostUnregister(const_cast<void *>((const void *)codes));
        }
        for (auto ptr : p_parts) {
            if (pin_flags[i++]) {
                ::hipHostUnregister(const_cast<void *>((const void *)ptr));
            }
        }
    };
    // Make sure unpin_memory is executed before exiting the current function,
    // even in case of exceptions.
    scoped_guard<decltype(unpin_memory)> sg_unpin_memory(unpin_memory);

    // Fetch how many gpus we will actually be using.
    // NOTE: this is ensured to be not greater than the value returned
    // by the CUDA api, due to the checks we do outside this function.
    // So, we can freely cast it around to unsigned and signed int as well.
    const auto ngpus = static_cast<unsigned>(split_indices.size() - 1u);

    // Create the arrays in managed memory that will hold the results.
    std::vector<arr_wrap<scoped_cu_array<F>, tree_nvecs_res<Q, NDim>>> res_arrs;
    std::vector<arr_wrap<F *, tree_nvecs_res<Q, NDim>>> res_ptrs;
    for (auto i = 0u; i < ngpus; ++i) {
        typename decltype(res_arrs)::value_type tmp_arrs;
        typename decltype(res_ptrs)::value_type tmp_ptrs;
        for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
            const auto a_size = boost::numeric_cast<std::size_t>(split_indices[i + 1u] - split_indices[i]);
            tmp_arrs.value[j] = scoped_cu_array<F>(a_size);
            tmp_ptrs.value[j] = tmp_arrs.value[j].get();
            // Tell the memory subsystem that we will use device i for this chunk of memory.
            cuda_mem_advise(tmp_ptrs.value[j], sizeof(F) * a_size, ::hipMemAdviseSetPreferredLocation,
                            static_cast<int>(i));
        }
        res_arrs.push_back(std::move(tmp_arrs));
        res_ptrs.push_back(std::move(tmp_ptrs));
    }

    // Copy over the tree data.
    scoped_cu_array<tree_node_t<NDim, F, UInt>> tree_arr(boost::numeric_cast<std::size_t>(tree_size));
    cuda_memcpy(tree_arr.get(), tree, sizeof(tree_node_t<NDim, F, UInt>) * tree_size, ::hipMemcpyDefault);
    const auto *tree_ptr = tree_arr.get();
    for (auto i = 0u; i < ngpus; ++i) {
        // This will be read-only memory.
        cuda_mem_advise(tree_ptr, sizeof(tree_node_t<NDim, F, UInt>) * tree_size, ::hipMemAdviseSetReadMostly,
                        static_cast<int>(i));
    }

    // Copy over the particles' data.
    arr_wrap<scoped_cu_array<F>, NDim + 1u> parts_arrs;
    arr_wrap<const F *, NDim + 1u> parts_ptrs;
    for (std::size_t j = 0; j < NDim + 1u; ++j) {
        parts_arrs.value[j] = scoped_cu_array<F>(boost::numeric_cast<std::size_t>(nparts));
        cuda_memcpy(parts_arrs.value[j].get(), p_parts[j], sizeof(F) * nparts, ::hipMemcpyDefault);
        parts_ptrs.value[j] = parts_arrs.value[j].get();
        for (auto i = 0u; i < ngpus; ++i) {
            cuda_mem_advise(parts_ptrs.value[j], sizeof(F) * nparts, ::hipMemAdviseSetReadMostly, static_cast<int>(i));
        }
    }

    // Copy over the codes.
    scoped_cu_array<UInt> codes_arr(boost::numeric_cast<std::size_t>(nparts));
    cuda_memcpy(codes_arr.get(), codes, sizeof(UInt) * nparts, ::hipMemcpyDefault);
    const auto *codes_ptr = codes_arr.get();
    for (auto i = 0u; i < ngpus; ++i) {
        cuda_mem_advise(codes_ptr, sizeof(UInt) * nparts, ::hipMemAdviseSetReadMostly, static_cast<int>(i));
    }

    // NOTE: not 100% sure this is necessary here, as the docs say that memory copy
    // functions have "mostly" synchronizing behaviour. However, hipMemAdvise() has async
    // behaviour, so it's probably better to enforce a barrier.
    for (auto i = 0u; i < ngpus; ++i) {
        cuda_set_device(static_cast<int>(i));
        cuda_device_synchronize();
    }

    // Prepare a vector of streams.
    std::vector<::hipStream_t> streams;

    // Function to wait on the streams and destroy them.
    auto wait_destroy_streams = [&streams]() noexcept
    {
        for (decltype(streams.size()) i = 0; i < streams.size(); ++i) {
            cuda_set_device(static_cast<int>(i));
            ::hipStreamSynchronize(streams[i]);
            ::hipStreamDestroy(streams[i]);
        }
    };
    // Make sure this is always called, even if exceptions are thrown.
    scoped_guard<decltype(wait_destroy_streams)> sg_wait_destroy_streams(wait_destroy_streams);

    // Actually create the streams.
    streams.reserve(boost::numeric_cast<decltype(streams.size())>(ngpus));
    for (auto i = 0u; i < ngpus; ++i) {
        cuda_set_device(static_cast<int>(i));
        ::hipStream_t tmp;
        cuda_stream_create(&tmp);
        streams.push_back(tmp);
    }

    // Run the computations on the devices.
    for (auto i = 0u; i < ngpus; ++i) {
        // Set the device.
        cuda_set_device(static_cast<int>(i));

        // Number of particles for which we will be
        // computing the accelerations/potentials for
        // this device.
        const auto loc_nparts = split_indices[i + 1u] - split_indices[i];

        // Run the kernel.
        if (loc_nparts > std::numeric_limits<unsigned>::max() - 31u) {
            throw std::overflow_error("Overflow in the number of particles to be assigned to a CUDA kernel");
        }
        acc_pot_kernel<Q, NDim, F, UInt><<<(loc_nparts + 31u) / 32u, 32u, 0, streams[i]>>>(
            res_ptrs[i], boost::numeric_cast<int>(split_indices[i]), boost::numeric_cast<int>(split_indices[i + 1u]),
            tree_ptr, boost::numeric_cast<int>(tree_size), parts_ptrs, codes_ptr, theta2, G, eps2);
    }

    // Write out the results.
    for (auto i = 0u; i < ngpus; ++i) {
        cuda_set_device(static_cast<int>(i));
        for (std::size_t j = 0; j < tree_nvecs_res<Q, NDim>; ++j) {
            cuda_memcpy_async(out[j] + split_indices[i], res_ptrs[i].value[j],
                              sizeof(F) * (split_indices[i + 1u] - split_indices[i]), ::hipMemcpyDefault, streams[i]);
        }
    }
}

// Explicit instantiations of the templates implemented above. We are going to use Boost.Preprocessor.
// It's gonna look ugly, but it will allow us to avoid a lot of typing.

// Define the values/types that we will use for the concrete instantiations.

// Only quadtrees and octrees for the moment.
#define RAKAU_CUDA_INST_DIM_SEQUENCE (2)(3)

// float and double only on the gpu.
#define RAKAU_CUDA_INST_FP_SEQUENCE (float)(double)

// 32/64bit types for the particle codes.
#define RAKAU_CUDA_INST_UINT_SEQUENCE (std::uint32_t)(std::uint64_t)

// Computation of accelerations, potentials or both.
#define RAKAU_CUDA_INST_Q_SEQUENCE (0)(1)(2)

// Macro for the instantiation of the main function. NDim, F, UInt and Q will be passed in
// as a sequence named Args (in that order).
#define RAKAU_CUDA_EXPLICIT_INST_FUN(r, Args)                                                                          \
    template void cuda_acc_pot_impl<BOOST_PP_SEQ_ELEM(3, Args), BOOST_PP_SEQ_ELEM(0, Args),                            \
                                    BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(2, Args)>(                           \
        const std::array<BOOST_PP_SEQ_ELEM(1, Args) *,                                                                 \
                         tree_nvecs_res<BOOST_PP_SEQ_ELEM(3, Args), BOOST_PP_SEQ_ELEM(0, Args)>> &,                    \
        const std::vector<tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>> &,                                                  \
        const tree_node_t<BOOST_PP_SEQ_ELEM(0, Args), BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(2, Args)> *,       \
        tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>,                                                                       \
        const std::array<const BOOST_PP_SEQ_ELEM(1, Args) *, BOOST_PP_SEQ_ELEM(0, Args) + 1u> &,                       \
        const BOOST_PP_SEQ_ELEM(2, Args) *, tree_size_t<BOOST_PP_SEQ_ELEM(1, Args)>, BOOST_PP_SEQ_ELEM(1, Args),       \
        BOOST_PP_SEQ_ELEM(1, Args), BOOST_PP_SEQ_ELEM(1, Args));

// Do the actual instantiation via a cartesian product over the sequences.
// clang-format off
BOOST_PP_SEQ_FOR_EACH_PRODUCT(RAKAU_CUDA_EXPLICIT_INST_FUN, (RAKAU_CUDA_INST_DIM_SEQUENCE)(RAKAU_CUDA_INST_FP_SEQUENCE)(RAKAU_CUDA_INST_UINT_SEQUENCE)(RAKAU_CUDA_INST_Q_SEQUENCE));
// clang-format on

} // namespace detail
} // namespace rakau
